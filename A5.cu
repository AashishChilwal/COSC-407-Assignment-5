#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

// CUDA kernel to initialize the array in parallel
__global__ void initializingArray(double *a, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculating dim i 
    if (i < n) {
        a[i] = (double)i / n;
    }
}

int main() {

    const int n = 10000000; // 10 million integer count
    double *d_a; // Device array
    double *a = (double*)malloc(n * sizeof(double)); // Host array dynamic allocation
    int block_size = 256; // Number of threads per block
    int num_blocks = (n + block_size - 1) / block_size; // Number of blocks

    // Record the starting time
    clock_t start_time = clock();

    // Allocating memory for the device array
    hipMalloc((void**)&d_a, n * sizeof(double));

    // Launching the CUDA kernel
    initializeArray<<<num_blocks, block_size>>>(d_a, n);

    // Copy the initialized array from the device to the host
    hipMemcpy(a, d_a, n * sizeof(double), hipMemcpyDeviceToHost);

    // Recording the ending time
    clock_t end_time = clock();

    // Calculating the elapsed time in seconds
    double elapsed_time = (double)(end_time - start_time) / CLOCKS_PER_SEC;

    // Printing the first 5 elements
    for (int i = 0; i < 5; i++) {
        printf("a[%d]: %.7f\n", i, a[i]);
    }

    // Printing the last 5 elements
    for (int i = n - 5; i < n; i++) {
        printf("a[%d]: %.7f\n", i, a[i]);
    }

    // Printing the elapsed time
    printf("Time taken: %.5f seconds\n", elapsed_time);

    // Freeing memory
    free(a);
    hipFree(d_a);

    return elapsed_time;
}
